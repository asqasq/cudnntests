#include <hipDNN.h>
#include <stdio.h>
#include <readmnist.h>

int create_simple_network(char *trainimg, char *trainlb, char *tstimg, char *tstlb)
{
    int res;
    int gpu_id = 0;

    // load data    
    struct mnist_img_desc traindesc;
    struct mnist_img_desc testdesc;

    //train
    res = read_train_mnist(trainimg, trainlb, &traindesc);
    if (res != 0) {
       printf("Error %d\n", res);
       return res;
    }

    //test
    res = read_train_mnist(tstimg, tstlb, &testdesc);
    if (res != 0) {
       printf("Error %d\n", res);
       return res;
    }


  
    hipSetDevice(gpu_id);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/128,
                                          /*channels=*/1,
                                          /*image_height=*/traindesc->rows,
                                          /*image_width=*/traindesc->cols));


