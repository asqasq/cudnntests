#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <stdio.h>
#include <readmnist.h>

#define checkCudaErrors(status) {         \
  if ((status) != 0) {                    \
    printf("\nCUDA error %d\n", status);  \
    exit(status);                         \
  }                                       \
};


#define checkCudnnErrors(status) {            \
  if ((status) != HIPDNN_STATUS_SUCCESS) {     \
    printf("\nCuDNN error %d: %s\n",          \
           status,                            \
           hipdnnGetErrorString(status));      \
    exit(status);                             \
  }                                           \
};



// Matrix mulitplication
// C = A * B
static void matrix_multiplication(float *A, int rowA, int colA, float *B, int rowB, int colB, float *C)
{
  // C = A * B
  for (int i = 0; i < rowA; i++) {
    for (int j = 0; j < colB; j++) {
      float sum = 0.0f;
      for (int e = 0; e < colA; e++) {
        sum += A[i * colA + e] * B[e * colB + j];
      }
      C[i * colB + j] = sum;
    }
  }
}

static void matrix_add(float *A, int rowA, int colA, float *B, int rowB, int colB, float *C)
{
    // C = A + B
    for (int i = 0; i < (rowA * colA); i++) {
        C[i] = A[i] + B[i];
    }
}

static void matrix_scaling(float v, float *A, int rowA, int colA)
{
    for (int i = 0; i < (rowA * colA); i++) {
        A[i] = v * A[i];
    }
}


static void forward_propagation(float *input,
                                float *weight1, float *bias1,
                                float *fc1out, float *fc1biasout,
                                float *weight2, float *bias2,
                                float *fc2out, float *fc2biasout)
{
    matrix_multiplication(input, 1, 784,
                          weight1, 784, 50,
                          fc1out);

    matrix_add(fc1out, 1, 50, bias1, 1, 50, fc1biasout);
    
    matrix_multiplication(fc1biasout, 1, 50,
                          weight1, 50, 10,
                          fc2out);

    matrix_add(fc2out, 1, 10, bias2, 1, 10, fc2biasout);
}


int create_simple_network(char *trainimg, char *trainlb, char *tstimg, char *tstlb)
{
    int res;

    // load data    
    struct mnist_img_desc traindesc;
    struct mnist_img_desc testdesc;

    //train
    res = read_train_mnist(trainimg, trainlb, &traindesc);
    if (res != 0) {
       printf("Error %d\n", res);
       return res;
    }

    //test
    res = read_train_mnist(tstimg, tstlb, &testdesc);
    if (res != 0) {
       printf("Error %d\n", res);
       return res;
    }

// 784, 50, 10

/*
1x784 * 784x50 = 1x50
1x50 + 1x50 = 1x50

1x50 * 50x10 = 1x10
1x10 * 1x10 = 1x10
*/

    float weights1[784 * 50];
    float bias1[50];
    float weights2[50 * 10];
    float bias2[10];
    
    float fc1out[50];
    float fc1biasout[50];
    
    float fc2out[10];
    float fc2biasout[10];
    
    
    
    forward_propagation(traindesc.databufferf,
                        weights1, bias1,
                        fc1out, fc1biasout,
                        weights2, bias2,
                        fc2out, fc2biasout);
    
    return 0;
}

#if 0
void bla() {
    int gpu_id = 0;

    hipSetDevice(gpu_id);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnTensorDescriptor_t input_descriptor;
    checkCudnnErrors(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCudnnErrors(hipdnnSetTensor4dDescriptor(input_descriptor,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/128,
                                          /*channels=*/1,
                                          /*image_height=*/traindesc->rows,
                                          /*image_width=*/traindesc->cols));
}


#include <stdio.h>
#include <math.h>

int main () {
   double x = 0;
  
   printf("The exponential value of %lf is %lf\n", x, exp(x));
   printf("The exponential value of %lf is %lf\n", x+1, exp(x+1));
   printf("The exponential value of %lf is %lf\n", x+2, exp(x+2));
   
   return(0);
}

#endif

